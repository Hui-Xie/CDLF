#include "hip/hip_runtime.h"
//
// Created by Hui Xie on 10/1/18.
// Copyright (c) 2018 Hui Xie. All rights reserved.
//
#include "DeviceKernels.h"

__global__ void deviceInitialize(float *pData, const long N, const float value) {
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    while (index < N){
        pData[index] = value;
        index += blockDim.x*gridDim.x;  //grid-stride loop
    }
}

__global__ void device2DMatrixProduct(float* pA, float* pB, float* pC, const long M,const long N, const long K){
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    long totalN  = M*N;
    while (index < N){
        pC[index] = 0.0f;



        index += blockDim.x*gridDim.x;  //grid-stride loop
    }
}