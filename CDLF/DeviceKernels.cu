#include "hip/hip_runtime.h"
//
// Created by Hui Xie on 10/1/18.
// Copyright (c) 2018 Hui Xie. All rights reserved.
//
#include "DeviceKernels.h"

__global__ void deviceInitialize(float *pData, const long N, const float value) {
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    while (index < N){
        pData[index] = value;
        index += blockDim.x*gridDim.x;  //grid-stride loop
    }
}

// C = A*B, where A has a size of M*K, B has a size of K*N, C will has a size of M*N
__global__ void device2DMatrixProduct(float* pA, float* pB, float* pC, const long M,const long N, const long K){
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    long totalN  = M*N;
    while (index < totalN){
        pC[index] = 0.0f;
        long m = index/N;
        long n = index%N;
        for (long i=0; i<K; ++i){
            pC[index] += pA[m*K+i]*pB[i*N+n];
        }
        index += blockDim.x*gridDim.x;  //grid-stride loop
    }
}

// C = A*d, where C has a length of N, d is a scalar
__global__ void deviceTensorMultiply(float* pA, const float d, float* pC, const long N){
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    while (index < N){
        pC[index] = pA[index] * d;
        index += blockDim.x*gridDim.x;
    }
}

// B = A', where B has a size M*N
__global__ void device2DMatrixTranspose(float* pA, float* pB, const long M, const long N){
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    long totalN  = M*N;
    while (index < totalN){
        long m = index/N;
        long n = index%N; //index = m*N+n
        pB[index] = pA[n*M+m];
        index += blockDim.x*gridDim.x;  //grid-stride loop
    }
}

// C = A+B, where C has a length of N
__global__ void deviceTensorAdd(float* pA, float* pB, float* pC, const long N){
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    while (index < N){
        pC[index] = pA[index] + pB[index];
        index += blockDim.x*gridDim.x;
    }
}

// C = A+d, where C has a length of N, d is a scalar
__global__ void deviceTensorAdd(float* pA, const float d, float* pC, const long N){
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    while (index < N){
        pC[index] = pA[index] + d;
        index += blockDim.x*gridDim.x;
    }
}

// C = A-B, where C has a length of N
__global__ void deviceTensorSubtraction(float* pA, float* pB, float* pC, const long N){
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    while (index < N){
        pC[index] = pA[index] - pB[index];
        index += blockDim.x*gridDim.x;
    }
}

// C = A-d, where C has a length of N, d is a scalar
__global__ void deviceTensorSubtraction(float* pA, const float d, float* pC, const long N){
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    while (index < N){
        pC[index] = pA[index] - d;
        index += blockDim.x*gridDim.x;
    }
}

// C = A/d, where C has a length of N, d is a scalar
__global__ void deviceTensorDivide(float* pA, const float d, float* pC, const long N){
    long index = threadIdx.x + blockIdx.x * blockDim.x;
    while (index < N){
        pC[index] = pA[index]/d;
        index += blockDim.x*gridDim.x;
    }
}