#include "hip/hip_runtime.h"
//
// Created by Hui Xie on 10/1/18.
// Copyright (c) 2018 Hui Xie. All rights reserved.
//
#include "DeviceKernels.h"

__global__ void deviceZeroInitialize(float *pData, const long N) {
    long index = threadIdx.x + blockIdx.x * blockDim.x;// + gridIdx.x * gridDim.x * blockDim.x;
    if (index < N) pData[index] = 0;
}

