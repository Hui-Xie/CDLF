#include "hip/hip_runtime.h"
//
// Created by Hui Xie on 10/10/18.
// Copyright (c) 2018 Hui Xie. All rights reserved.

//

#include "LayerKernels.h"

__global__ void deviceSigmoidDerivative(float* pX, float* pdY, const int k, float* pdX, const long N){
    long i = threadIdx.x + blockIdx.x * blockDim.x; //i: thread index
    while (i < N){
        float  expx = exp(pX[i]);
        pdX[i] += pdY[i]*k*expx/pow(1+expx,2);
        i += blockDim.x*gridDim.x;
    }
}