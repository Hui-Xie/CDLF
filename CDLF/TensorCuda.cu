//
// Created by Hui Xie on 10/2/18.
// Copyright (c) 2018 Hui Xie. All rights reserved.
//

#include "TensorCuda.h"
#include "DeviceKernels.h"
#include "GPUAttr.h"

void cudaInitialize(float* m_data, const long N, const float value){
    deviceInitialize<<<g_blocksPerGrid, g_maxThreadsPerBlock>>>(m_data, N, value);
    hipDeviceSynchronize();
}

// C = A*B, where A has a size of M*K, B has a size of k*N, C will has a size of M*N
void cuda2DMatrixProduct(float* pA, float* pB, float* pC, const long M,const long N, const long K){
    device2DMatrixProduct<<<g_blocksPerGrid, g_maxThreadsPerBlock>>>(pA,pB,pB, M, N, K);
    hipDeviceSynchronize();
}