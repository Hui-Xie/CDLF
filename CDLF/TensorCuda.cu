//
// Created by Hui Xie on 10/2/18.
// Copyright (c) 2018 Hui Xie. All rights reserved.
//

#include "TensorCuda.h"
#include "DeviceKernels.h"
#include "GPUAttr.h"

void cudaInitialize(float* m_data, const long N, const float value){
    deviceInitialize<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(m_data, N, value);
    hipDeviceSynchronize();
}

// C = A*B, where A has a size of M*K, B has a size of K*N, C will has a size of M*N
void cuda2DMatrixProduct(float* pA, float* pB, float* pC, const long M,const long N, const long K){
    device2DMatrixProduct<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA,pB,pC, M, N, K);
    hipDeviceSynchronize();
}

// B = A', where B has a size M*N
void cuda2DMatrixTranspose(float* pA, float* pB, const long M, const long N){
    device2DMatrixTranspose<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA,pB,M, N);
    hipDeviceSynchronize();

}

// C = A+B, where C has a length of N
void cudaTensorAdd(float* pA, float* pB, float* pC, const long N){
    deviceTensorAdd<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA, pB, pC, N);
    hipDeviceSynchronize();
}

// C = A+d, where C has a length of N, d is scalar
void cudaTensorAdd(float* pA, const float d, float* pC, const long N){
    deviceTensorAdd<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA, d, pC, N);
    hipDeviceSynchronize();
}

// C = A-B, where C has a length of N
void cudaTensorSubtraction(float* pA, float* pB, float* pC, const long N){
    deviceTensorSubtraction<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA, pB, pC, N);
    hipDeviceSynchronize();
}

// C = A-d, where C has a length of N, d is scalar
void cudaTensorSubtraction(float* pA, const float d, float* pC, const long N){
    deviceTensorSubtraction<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA, d, pC, N);
    hipDeviceSynchronize();
}

// C = A/d, where C has a length of N, d is scalar
void cudaTensorDivide(float* pA, const float d, float* pC, const long N){
    deviceTensorDivide<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA, d, pC, N);
    hipDeviceSynchronize();
}