//
// Created by Hui Xie on 10/10/18.
// Copyright (c) 2019 Hui Xie. All rights reserved.

//

#include "LayerCuda.h"
#include "LayerKernels.h"
#include "hip/hip_runtime.h"
#include "GPUAttr.h"

/*
void cudaSigmoidDerivative(const float* __restrict__  pX, const float* __restrict__  pdY, float* pdX, const int k, const int N){
    deviceSigmoidDerivative<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pdY, pdX, k, N);
    hipDeviceSynchronize();
}

void cudaSigmoid(const float* __restrict__  pX, float* pY, const int k, const int N){
    deviceSigmoid<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pY, k,N);
    hipDeviceSynchronize();
}

void cudaCrossEntropyGradient(const float* __restrict__  pX, const float* __restrict__  pGTX, float* pdX, const float epsilon, const int N){
    deviceCrossEntropyGradient<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pGTX, pdX, epsilon,N);
    hipDeviceSynchronize();
}

//C = A where A and C has different value type
void cudaElementCopy(const unsigned char* __restrict__ pA,float* pC, const int N){
    deviceElementCopy<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA, pC, N);
    hipDeviceSynchronize();
}

//C = A if A>=0; C =0 else
void cudaRelu(const float* __restrict__  pA,float* pC, const int N){
    deviceRelu<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA, pC, N);
    hipDeviceSynchronize();
}

// dL/dx = dL/dy * dy/dx = dL/dy if X>=0; 0 if X < 0
void cudaReluDerivative(const float* __restrict__  pX,const float* __restrict__  pdY, float* pdX, const int N){
    deviceReluDerivative<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pdY, pdX, N);
    hipDeviceSynchronize();
}

void cudaSoftmax(const float* __restrict__  pX, float* pY, const int nSoftmax, const int N){
    deviceSoftmax<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pY, nSoftmax, N);
    hipDeviceSynchronize();
}

void cudaSoftmaxDerivative(const float* __restrict__  pX,const float* __restrict__  pdY, float* pdX, const int nSoftmax, const int N){
    deviceSoftmaxDerivative<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pdY, pdX, nSoftmax, N);
    hipDeviceSynchronize();
}

//C = A*F in convolution
void cudaConvLayerForward(const float* pA, const int* pADimsSpan, const float* pF, const int* pFDimsSpan, const int filterSize, const int NFilter,
                          const int stride, float* pC, const int* pCDimsSpan, const int* pNonZeroIndex, const int CDimsSize, const int N){
    int  byteLengthSharedMem = CDimsSize*sizeof(int)+ filterSize*sizeof(int);
    deviceConvLayerForward<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock/2, byteLengthSharedMem>>>
                         (pA,pADimsSpan,pF,pFDimsSpan, filterSize, NFilter, stride, pC, pCDimsSpan,pNonZeroIndex,CDimsSize, N);
    hipDeviceSynchronize();
}
*/