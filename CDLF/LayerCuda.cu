//
// Created by Hui Xie on 10/10/18.
// Copyright (c) 2018 Hui Xie. All rights reserved.

//

#include "LayerCuda.h"
#include "LayerKernels.h"
#include "hip/hip_runtime.h"
#include "GPUAttr.h"

void cudaSigmoidDerivative(float* pX, float* pdY, const int k, float* pdX, const long N){
    deviceSigmoidDerivative<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pdY, k,pdX, N);
    hipDeviceSynchronize();
}

void cudaSigmoid(float* pX, float* pY, const int k, const long N){
    deviceSigmoid<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pY, k,N);
    hipDeviceSynchronize();
}

void cudaCrossEntropyGradient(float* pX, float* pGTX, float* pdX, const float epsilon, const long N){
    deviceCrossEntropyGradient<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pGTX, pdX, epsilon,N);
    hipDeviceSynchronize();
}

//C = A where A and C has different value type
void cudaElementCopy(unsigned char* pA,float* pC, const long N){
    deviceElementCopy<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA, pC, N);
    hipDeviceSynchronize();
}

//C = A if A>=0; C =0 else
void cudaRelu(float* pA,float* pC, const long N){
    deviceRelu<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pA, pC, N);
    hipDeviceSynchronize();
}

// dL/dx = dL/dy * dy/dx = dL/dy if X>=0; 0 if X < 0
void cudaReluDerivative(float* pX,float* pdY, float* pdX, const long N){
    deviceReluDerivative<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pdY, pdX, N);
    hipDeviceSynchronize();
}