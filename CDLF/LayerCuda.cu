//
// Created by Hui Xie on 10/10/18.
// Copyright (c) 2018 Hui Xie. All rights reserved.

//

#include "LayerCuda.h"
#include "LayerKernels.h"
#include "hip/hip_runtime.h"

void cudaSigmoidDerivative(float* pX, float* pdY, const int k, float* pdX, const long N){
    deviceSigmoidDerivative<<<GPUAttr::m_blocksPerGrid, GPUAttr::m_maxThreadsPerBlock>>>(pX, pdY, k,pdX, N);
    hipDeviceSynchronize();
}
